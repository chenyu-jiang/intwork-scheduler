#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2020 by Contributors
 * \file relu_lib.cu
 * \brief simple custom relu operator implemented using CUDA function
 */

#include <iostream>
#include "lib_api.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        out[tid] = in[tid] > 0 ? in[tid] : 0;
}

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        ingrad[tid] = indata[tid] > 0 ? 1 * outgrad[tid] : 0;
}

MXReturnValue forwardCPU(std::map<std::string, std::string> attrs,
                         std::vector<MXTensor> inputs,
                         std::vector<MXTensor> outputs,
                         OpResource res) {
    float* in_data = inputs[0].data<float>();
    float* out_data = outputs[0].data<float>();
    for (int i=0; i<inputs[0].size(); i++) {
        out_data[i] = in_data[i] > 0 ? in_data[i] : 0;
    }
    return MX_SUCCESS;
}

MXReturnValue backwardCPU(std::map<std::string, std::string> attrs,
                          std::vector<MXTensor> inputs,
                          std::vector<MXTensor> outputs,
                          OpResource res) {
    float* out_grad = inputs[0].data<float>();
    float* in_data = inputs[1].data<float>();
    float* in_grad = outputs[0].data<float>();
    for (int i=0; i<inputs[1].size(); i++) {
        in_grad[i] = in_data[i] > 0 ? 1 * out_grad[i] : 0;
    }
    return MX_SUCCESS;
}

MXReturnValue forwardGPU(std::map<std::string, std::string> attrs,
                         std::vector<MXTensor> inputs,
                         std::vector<MXTensor> outputs,
                         OpResource res) {
    float* in_data = inputs[0].data<float>();
    float* out_data = outputs[0].data<float>();

    mx_stream_t hip_stream = res.get_cuda_stream();
    int64_t N = inputs[0].size();
    int block = 256;
    int grid = (N + (block - 1)) / block;
    relu_gpu_forward<<<grid,block,0,hip_stream>>>(out_data, in_data, N);

    return MX_SUCCESS;
}

MXReturnValue backwardGPU(std::map<std::string, std::string> attrs,
                          std::vector<MXTensor> inputs,
                          std::vector<MXTensor> outputs,
                          OpResource res) {
    float* out_grad = inputs[0].data<float>();
    float* in_data = inputs[1].data<float>();
    float* in_grad = outputs[0].data<float>();

    mx_stream_t hip_stream = res.get_cuda_stream();
    int64_t N = inputs[0].size();
    int block = 256;
    int grid = (N + (block - 1)) / block;
    relu_gpu_backward<<<grid,block,0,hip_stream>>>(in_grad, out_grad, in_data, N);

    return MX_SUCCESS;
}

MXReturnValue parseAttrs(std::map<std::string, std::string> attrs, int* num_in, int* num_out) {
    *num_in = 1;
    *num_out = 1;
    return MX_SUCCESS;
}

MXReturnValue inferType(std::map<std::string, std::string> attrs,
                        std::vector<int> &intypes,
                        std::vector<int> &outtypes) {
    outtypes[0] = intypes[0];
    return MX_SUCCESS;
}

MXReturnValue inferShape(std::map<std::string, std::string> attrs,
                         std::vector<std::vector<unsigned int>> &inshapes,
                         std::vector<std::vector<unsigned int>> &outshapes) {
    outshapes[0] = inshapes[0];
    return MX_SUCCESS;
}

REGISTER_OP(my_relu)
.setParseAttrs(parseAttrs)
.setInferType(inferType)
.setInferShape(inferShape)
.setForward(forwardCPU, "cpu")
.setForward(forwardGPU, "gpu")
.setBackward(backwardCPU, "cpu")
.setBackward(backwardGPU, "gpu");

class MyStatefulReluCPU : public CustomStatefulOp {
public:
    explicit MyStatefulReluCPU() {}
    MXReturnValue Forward(std::vector<MXTensor> inputs,
                          std::vector<MXTensor> outputs,
                          OpResource op_res) {
        std::map<std::string, std::string> attrs;
        return forwardCPU(attrs, inputs, outputs, op_res);
    }
    MXReturnValue Backward(std::vector<MXTensor> inputs,
                           std::vector<MXTensor> outputs,
                           OpResource op_res) {
        std::map<std::string, std::string> attrs;
        return backwardCPU(attrs, inputs, outputs, op_res);
    }
    ~MyStatefulReluCPU() {}
};

class MyStatefulReluGPU : public CustomStatefulOp {
public:
    explicit MyStatefulReluGPU() {}
    MXReturnValue Forward(std::vector<MXTensor> inputs,
                          std::vector<MXTensor> outputs,
                          OpResource op_res) {
        std::map<std::string, std::string> attrs;
        return forwardGPU(attrs, inputs, outputs, op_res);
    }
    MXReturnValue Backward(std::vector<MXTensor> inputs,
                           std::vector<MXTensor> outputs,
                           OpResource op_res) {
        std::map<std::string, std::string> attrs;
        return backwardGPU(attrs, inputs, outputs, op_res);
    }
    ~MyStatefulReluGPU() {}
};

MXReturnValue createOpStateCPU(std::map<std::string, std::string> attrs,
                               CustomStatefulOp** op_inst) {
    *op_inst = new MyStatefulReluCPU();
    return MX_SUCCESS;
}

MXReturnValue createOpStateGPU(std::map<std::string, std::string> attrs,
                               CustomStatefulOp** op_inst) {
    *op_inst = new MyStatefulReluGPU();
    return MX_SUCCESS;
}

REGISTER_OP(my_state_relu)
.setParseAttrs(parseAttrs)
.setInferType(inferType)
.setInferShape(inferShape)
.setCreateOpState(createOpStateCPU, "cpu")
.setCreateOpState(createOpStateGPU, "gpu");

MXReturnValue initialize(int version) {
    if (version >= 10400) {
        std::cout << "MXNet version " << version << " supported" << std::endl;
        return MX_SUCCESS;
    } else {
        std::cout << "MXNet version " << version << " not supported" << std::endl;
        return MX_FAIL;
    }
}
